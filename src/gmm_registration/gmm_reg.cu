#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include "gmm_reg.h"
#include "../common/utilities.h"
#include "gmm_reg_kernels.h"


/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Size of the starting area in simulation space. */
#define scene_scale 0.1


GMMRegistration::GMMRegistration(int K) {
	this->numComponents = K;
}

void GMMRegistration::initSimulation(int numSrc, glm::vec3* srcPc, int numTarget, glm::vec3* targetPc) {
	// allocate src pc and src trans pc
	this->numSrcPc = numSrc;
	hipMalloc((void**)&(this->dev_srcPc), numSrc * sizeof(glm::vec3));
	hipMalloc((void**)&(this->dev_srcTransPc), numSrc * sizeof(glm::vec3));

	// allocate target pc
	this->numTargetPc = numTarget;
	hipMalloc((void**)&(this->dev_targetPc), numTarget * sizeof(glm::vec3));

	// allocate src GMM mu & target GMM mu
	hipMalloc((void**)&(this->dev_srcMu), numComponents * sizeof(glm::vec3));
	hipMalloc((void**)&(this->dev_targetMu), numComponents * sizeof(glm::vec3));
	// allocate src GMM psi & target GMM psi
	hipMalloc((void**)&(this->dev_srcPsi), numComponents * sizeof(float));
	hipMalloc((void**)&(this->dev_targetPsi), numComponents * sizeof(float));

	checkCUDAErrorWithLine("hipMalloc failed!", __LINE__);

	hipMemcpy(dev_srcPc, srcPc, numSrc * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipMemcpy(dev_targetPc, targetPc, numTarget * sizeof(glm::vec3), hipMemcpyHostToDevice);

	checkCUDAErrorWithLine("cudaMecmpy failed!", __LINE__);

}

void GMMRegistration::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numSrcPc + numTargetPc + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO<< <fullBlocksPerGrid, blockSize >>>(numSrcPc, dev_srcPc, numTargetPc, dev_targetPc, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO<< <fullBlocksPerGrid, blockSize >>>(numSrcPc, numTargetPc, vbodptr_velocities, scene_scale);
	checkCUDAErrorWithLine("copyBoidsToVBO failed!", __LINE__);
	hipDeviceSynchronize();
}

void GMMRegistration::pointCloudRegisterGPU(float dt) {

}


void GMMRegistration::endSimulation() {


	hipFree(dev_srcPc);
	hipFree(dev_targetPc);
	hipFree(dev_srcTransPc);

	hipFree(dev_srcMu);
	hipFree(dev_targetMu);

	hipFree(dev_srcPsi);
	hipFree(dev_targetPsi);

	checkCUDAErrorWithLine("hipFree failed!", __LINE__);
}